#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <ctime>
#include "kernel.cu.h"
int main()
{
    std::clock_t start;
    
    unsigned int num_elems = (1 << 16);
    unsigned int* h_in = new unsigned int[num_elems];
    unsigned int* h_in_rand = new unsigned int[num_elems];
    unsigned int* h_out_gpu = new unsigned int[num_elems];

    for (int j = 0; j < num_elems; j++)
    {
        h_in[j] = (num_elems - 1) - j;
        h_in_rand[j] = rand() % num_elems;
    }

    unsigned int* d_in;
    unsigned int* d_out;
    hipMalloc(&d_in, sizeof(unsigned int) * num_elems);
    hipMalloc(&d_out, sizeof(unsigned int) * num_elems);
    hipMemcpy(d_in, h_in_rand, sizeof(unsigned int) * num_elems, hipMemcpyHostToDevice);
    start = std::clock();
    radix_sort(d_out, d_in, num_elems);
    double gpu_duration = (std::clock() - start) / (double)CLOCKS_PER_SEC;
    std::cout << "GPU time: " << gpu_duration << " s" << std::endl;
    hipMemcpy(h_out_gpu, d_out, sizeof(unsigned int) * num_elems, hipMemcpyDeviceToHost);
    hipFree(d_out);
    hipFree(d_in);

}