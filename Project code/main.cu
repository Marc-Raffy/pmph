#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>
#include <iomanip>
#include <ctime>
#include "kernel.cuh"

void cpu_sort(unsigned int* h_out, unsigned int* h_in, size_t len)
{
    for (int i = 0; i < len; ++i)
    {
        h_out[i] = h_in[i];
    }
    std::sort(h_out, h_out + len);
}

int main()
{
    std::clock_t start;
    unsigned int elems_size = (1 << 28);
    int num_elems = (1 << 40);
    unsigned int* h_in = new unsigned int[num_elems];
    unsigned int* h_in_rand = new unsigned int[num_elems];
    unsigned int* h_out_gpu = new unsigned int[num_elems];
    unsigned int* h_out_cpu = new unsigned int[num_elems];
    for (int j = 0; j < num_elems; j++)
    {
        h_in[j] = (elems_size - 1) - j;
        h_in_rand[j] = rand() % elems_size;
    }

    /*start = std::clock();
    cpu_sort(h_out_cpu, h_in_rand, num_elems);  
    double cpu_duration = (std::clock() - start) / (double)CLOCKS_PER_SEC;
    std::cout << "CPU time: " << cpu_duration << " s" << std::endl;*/

    unsigned int* d_in;
    unsigned int* d_out;
    hipMalloc(&d_in, sizeof(unsigned int) * num_elems);
    hipMalloc(&d_out, sizeof(unsigned int) * num_elems);
    hipMemcpy(d_in, h_in_rand, sizeof(unsigned int) * num_elems, hipMemcpyHostToDevice);

    start = std::clock();
    radix_sort(d_out, d_in, num_elems);
    double gpu_duration = (std::clock() - start) / (double)CLOCKS_PER_SEC;
    
    std::cout << "GPU time: " << gpu_duration << " s" << std::endl;
    //bool match = true;
    hipMemcpy(h_out_gpu, d_out, sizeof(unsigned int) * num_elems, hipMemcpyDeviceToHost);
    /*for (int i = 0; i < num_elems; ++i)
    {
        if (h_out_cpu[i] != h_out_gpu[i])
        {
            match = false;
        }
    }*/
    //std::cout << "Match: " << match << std::endl;
    /*for (int i = 0; i < num_elems; i++)
    {
        std::cout <<  h_out_gpu[i] << std::endl;
    }*/
    

    hipMemcpy(h_out_gpu, d_out, sizeof(unsigned int) * num_elems, hipMemcpyDeviceToHost);
    hipFree(d_out);
    hipFree(d_in);

}