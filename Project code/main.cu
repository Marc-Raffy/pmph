#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>
#include <iomanip>
#include <ctime>
#include "kernel.cuh"

void cpu_sort(unsigned int* h_out, unsigned int* h_in, size_t len)
{
    for (int i = 0; i < len; ++i)
    {
        h_out[i] = h_in[i];
    }
    std::sort(h_out, h_out + len);
}

int main()
{
    std::clock_t start;
    for(int shift = 16; shift <31; shift++){
    
        unsigned int num_elems = (1 << shift);
        unsigned int* h_in = new unsigned int[num_elems];
        unsigned int* h_in_rand = new unsigned int[num_elems];
        unsigned int* h_out_gpu = new unsigned int[num_elems];
        unsigned int* h_out_cpu = new unsigned int[num_elems];
        for (int j = 0; j < num_elems; j++)
        {
            h_in[j] = (num_elems - 1) - j;
            h_in_rand[j] = rand() % num_elems;
        }

        /*start = std::clock();
        cpu_sort(h_out_cpu, h_in_rand, num_elems);  
        double cpu_duration = (std::clock() - start) / (double)CLOCKS_PER_SEC;
        std::cout << "CPU time: " << cpu_duration << " s" << std::endl;*/

        

        unsigned int* d_in;
        unsigned int* d_out;
        hipMalloc(&d_in, sizeof(unsigned int) * num_elems);
        hipMalloc(&d_out, sizeof(unsigned int) * num_elems);
        hipMemcpy(d_in, h_in_rand, sizeof(unsigned int) * num_elems, hipMemcpyHostToDevice);

        /*start = std::clock();
        radix_sort(d_out, d_in, num_elems);
        double gpu_duration = (std::clock() - start) / (double)CLOCKS_PER_SEC;
        
        std::cout << "GPU time: " << gpu_duration << " s" << std::endl;*/
        
        
        start = std::clock();
        void     *d_temp_storage = NULL;
        size_t   temp_storage_bytes = 0;
        hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_in, d_out, num_elems);
        // Allocate temporary storage
        hipMalloc(&d_temp_storage, temp_storage_bytes);
        // Run sorting operation
        hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_in, d_out, num_elems);
        double cub_duration = (std::clock() - start) / (double)CLOCKS_PER_SEC;
        
        std::cout << "CUB time: " << cub_duration << " s" << std::endl;

        bool match = true;
        hipMemcpy(h_out_gpu, d_out, sizeof(unsigned int) * num_elems, hipMemcpyDeviceToHost);
        /*for (int i = 0; i < num_elems; ++i)
        {
            if (h_out_cpu[i] != h_out_gpu[i])
            {
                match = false;
            }
        }*/

        std::cout << "Match: " << match << std::endl;
        hipMemcpy(h_out_gpu, d_out, sizeof(unsigned int) * num_elems, hipMemcpyDeviceToHost);
        hipFree(d_out);
        hipFree(d_in);
    }
}