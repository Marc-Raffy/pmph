#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>

__global__ void squareKernel(float* d_in, float *d_out) {
    const unsigned int lid = threadIdx.x;
    const unsigned int gid = blockIdx.x*blockDim.x + lid;
    d_out[gid] = d_in[gid]*d_in[gid];
}

void cpu_function(float* array_input, float* array_output, int array_size){
	for (int i = 0; i < array_size; i++)
	{
		array_output[i] = pow((array_input[i]/array_input[i]-2.3), 3);
	}
}

int main(int argc, char** argv){
	unsigned int N = 512;
    unsigned int mem_size = N*sizeof(float);

    // allocate host memory for GPU function
    float* h_in  = (float*) malloc(mem_size);
    float* h_out = (float*) malloc(mem_size);
	//allocate host memory for CPU function
	//float* array_input  = (float*) malloc(mem_size);
    //float* array_output = (float*) malloc(mem_size);

    // initialize the memory
    for(unsigned int i=1; i<=N; ++i) {
        h_in[i] = (float)i;
		//array_input[i] = (float)i;
    }
    //allocate device memory
	float* d_in;
    float* d_out;
    hipMalloc((void**)&d_in,  mem_size);
    hipMalloc((void**)&d_out, mem_size);

    hipMemcpy(d_in, h_in, mem_size, hipMemcpyHostToDevice);

    squareKernel<<< 1, 128>>>(d_in, d_out);

    hipMemcpy(h_out, d_out, mem_size, hipMemcpyDeviceToHost);

    // print result
    for(unsigned int i=0; i<N; ++i) printf("%.6f\n", h_out[i]);

    //free(array_input); //free(array_output);
    free(h_in);        free(h_out);
    hipFree(d_in);    hipFree(d_out);

}