#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>

__global__ void squareKernel(float* d_in, float *d_out, int N) {
    const unsigned int lid = threadIdx.x;
    const unsigned int gid = blockIdx.x*blockDim.x + lid;
    if(gid < N)
    {
        d_out[gid] = pow(d_in[gid]/(d_in[gid]-2.3),3);
    }
}

void cpu_function(float* array_input, float* array_output, int array_size){
	for (int i = 0; i < array_size; i++)
	{
		array_output[i] = pow((array_input[i]/array_input[i]-2.3), 3);
	}
}

int main(int argc, char** argv){
	unsigned int N = 753412;
    unsigned int mem_size = N*sizeof(float);
    unsigned int block_size = 256;
    unsigned int num_blocks = ((N + (block_size - 1) / block_size));
    
    // allocate host memory for GPU function
    float* h_in  = (float*) malloc(mem_size);
    float* h_out = (float*) malloc(mem_size);
	//allocate host memory for CPU function
	float* array_input  = (float*) malloc(mem_size);
    float* array_output = (float*) malloc(mem_size);
    // initialize the memory
    for(unsigned int i=1; i<N; ++i) {
        h_in[i] = (float)i;
		array_input[i] = (float)i;
    }
    //runs CPU function
    cpu_function(array_input, array_output, N);

    //allocate device memory
	float* d_in;
    float* d_out;
    hipMalloc((void**)&d_in,  mem_size);
    hipMalloc((void**)&d_out, mem_size);

    hipMemcpy(d_in, h_in, mem_size, hipMemcpyHostToDevice);

    squareKernel<<< num_blocks, block_size>>>(d_in, d_out, N);

    hipMemcpy(h_out, d_out, mem_size, hipMemcpyDeviceToHost);

    // check if results match
    int flag = 0;
    for(unsigned int i=1; i<N; ++i) {
        if(array_output[i] != h_out[i]){
            flag++;
        }
    }
    printf("%f Number of elements that do not match", flag);

    free(array_input); free(array_output);
    free(h_in);        free(h_out);
    hipFree(d_in);    hipFree(d_out);

}